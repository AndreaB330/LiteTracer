#include "hip/hip_runtime.h"
#include "litetracer.h"
#include ""
#include "hiprand.h"
#include "hiprand/hiprand_kernel.h"
#include <algorithm>
using std::swap;
#define RANDOM_PIXELS

int polygons_number = 0;
int polygons_allocated = 0;
Polygon * polygons_d;
float3 * buffer_d;
float3* accumulator_d;

float3 camera = { 2,4,5.5f };
float3 direction = normalize(make_float3(0, 1.8f, 0) - camera);

__constant__ float3 world_up = { 0,1,0 };
__constant__ float cam_to_screen = 0.9f;//~fov
__constant__ float scale = 1.0f / (width < height ? width : height);
__constant__ float EPS = 1e-6f;
__constant__ float INF = 1e6f;

__constant__ float magic_factor = 1.4f;//magic factor for light intensity tuning

__constant__ int max_bounces = 18;

void reallocate()
{
	Polygon * polygons = new Polygon[polygons_number];
	hipMemcpy(polygons, polygons_d, polygons_number * sizeof(Polygon), hipMemcpyDeviceToHost);
	hipFree(polygons_d);
	hipMalloc(&polygons_d, polygons_allocated * sizeof(Polygon));
	hipMemcpy(polygons_d, polygons, polygons_number * sizeof(Polygon), hipMemcpyHostToDevice);
	delete polygons;
}

__host__ __device__ float3 vec3(float x, float y, float z)
{
	return make_float3(x, y, z);
}

void add_polygon(float3 a, float3 b, float3 c, Material material, bool is_plane)
{
	if (polygons_number >= polygons_allocated)
	{
		polygons_allocated++;
		polygons_allocated *= 1.3;//in advance, std::vector-style
		reallocate();
	}
	Polygon polygon = { a,b,c,material,is_plane, normalize(cross(b - a,c - a)) };
	hipMemcpy(polygons_d + polygons_number++, &polygon, sizeof(Polygon), hipMemcpyHostToDevice);
}

void add_cube(float3 center, float3 sizes, Material material, float angle)
{
	float3 verticies[8];
	for (int i = 0; i < 8; i++)
	{
		verticies[i].x = (2 * (bool)(i & 1) - 1)*sizes.x * 0.5f;
		verticies[i].y = (2 * (bool)(i & 2) - 1)*sizes.y * 0.5f;
		verticies[i].z = (2 * (bool)(i & 4) - 1)*sizes.z * 0.5f;
		verticies[i] = rotate_y(verticies[i], angle);
		verticies[i] += center;
	}
	//verts in reversed order
	int3 polygon_verts[12] = {
		{0,1,3},//front
		{0,3,2},//front
		{1,5,7},//right
		{1,7,3},//right
		{4,0,2},//left
		{4,2,6},//left
		{2,3,7},//top
		{2,7,6},//top
		{4,5,1},//bottom
		{4,1,0},//bottom
		{5,4,6},//back
		{5,6,7}//back
	};
	for (int i = 0; i < 12; i++)
	{
		float3 a = verticies[polygon_verts[i].z];
		float3 b = verticies[polygon_verts[i].y];
		float3 c = verticies[polygon_verts[i].x];
		add_polygon(a, b, c, material, false);
	}
}

void build_room(float3 sizes)
{
	float3 verticies[8];
	for (int i = 0; i < 8; i++)
	{
		verticies[i].x = (2 * (bool)(i & 1) - 1)*sizes.x  * 0.5f;
		verticies[i].y = (2 * (bool)(i & 2) - 0)*sizes.y  * 0.5f;
		verticies[i].z = (2 * (bool)(i & 4) - 1)*sizes.z  * 0.5f;
	}
	int3 wall_verts[6] = {
		{ 0,1,3 },//front
		{ 1,5,7 },//right
		{ 4,0,2 },//left
		{ 2,3,7 },//top
		{ 4,5,1 },//bottom
		{ 5,4,6 },//back
	};
	Material materials[6] = {
		mirror_ideal,
		basic_yellow,
		basic_blue,
		light_white,
		basic_white,
		basic_white,
	};
	for (int i = 0; i < 6; i++)
	{
		float3 a = verticies[wall_verts[i].x];
		float3 b = verticies[wall_verts[i].y];
		float3 c = verticies[wall_verts[i].z];
		add_polygon(a, b, c, materials[i], 1);
	}
}

inline __host__ __device__ float intersect(Polygon poly, float3 origin, float3 direction)
{
	//returns distance from origin to intersection point on poligon
	float distance = -dot((origin - poly.a), poly.norm) / dot(direction, poly.norm);
	if (distance < EPS || (dot(poly.norm, direction) > EPS && poly.is_plane))
		return -INF;
	if (poly.is_plane)
		return distance;
	float3 hit = origin + direction*distance;
	if (dot(cross(poly.b - poly.a, hit - poly.a), poly.norm) > EPS &&
		dot(cross(poly.c - poly.b, hit - poly.b), poly.norm) > EPS &&
		dot(cross(poly.a - poly.c, hit - poly.c), poly.norm) > EPS)
		return distance;
	return -INF;
}

inline __device__ float3 randomCosin(float3 n, hiprandState & state)
{
	//cosine weighted random direction
	float r1 = 2 * PI * hiprand_uniform(&state);
	float r2 = hiprand_uniform(&state);
	float r2s = sqrtf(r2);
	float3 u = normalize(cross((fabs(n.x) > .1 ? make_float3(0, 1, 0) : make_float3(1, 0, 0)), n));
	float3 v = cross(n, u);
	return normalize(u*cos(r1)*r2s + v*sin(r1)*r2s + n*sqrtf(1 - r2));
}

__global__ void render_kernel(int iteration, int seed, int polygons_number, Polygon * polygons, float3 * accumulator, float3 * output, float3 camera, float3 cam_direction)
{
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if (idx >= image_size) return;
	hiprandState state;
	hiprand_init(seed * idx + iteration * idx + iteration + seed, 0, 0, &state);
#ifdef RANDOM_PIXELS
	//Anti-aliasing effect
	float x = scale*(idx % width - width / 2 + hiprand_uniform(&state));
	float y = scale*(idx / width - height / 2 + hiprand_uniform(&state));
#else
	float x = scale*(idx % width - width / 2);
	float y = scale*(idx / width - height / 2);
#endif
	float3 right = cross(cam_direction, world_up);
	float3 up = cross(right, cam_direction);
	float3 ray_origin = camera;
	float3 ray_direction = normalize(cam_to_screen*cam_direction + x*right + y*up);

	float3 mask = { 1.0f,1.0f,1.0f };//color mask
	float3 accumulated = { 0,0,0 };//accumulate color while a ray bounces
	for (int bounce = 0; bounce < max_bounces; bounce++)
	{
		float nearest = INF;
		int nearest_id = -1;
		for (int i = 0; i < polygons_number; i++)
		{
			float distance = intersect(polygons[i], ray_origin, ray_direction);
			if (distance > EPS && distance < nearest)
			{
				nearest = distance;
				nearest_id = i;
			}
		}
		if (nearest_id == -1) break;
		float3 n = polygons[nearest_id].norm;
		ray_origin += ray_direction*nearest;
		float3 random_direction = randomCosin(n, state);
		float3 reflected_direction = reflect(ray_direction, n);
		float reflection_factor = polygons[nearest_id].material.reflect;


		float refraction_index = polygons[nearest_id].material.refract;
		//if refractive material
		if (refraction_index > EPS)
		{
			//if ray goes from inside
			if (dot(n, ray_direction) > EPS)
				refraction_index = 1.0f / refraction_index;
			else
				n *= -1;
			float nsin = length(cross(n, ray_direction)) * refraction_index;//sin of outcoming ray
			if (fabsf(nsin) < 1.0f)
			{
				float ntg = nsin / sqrtf(1.0f - nsin*nsin);//tan of outcoming ray
				ray_direction = normalize(n + ntg*normalize(cross(cross(n, ray_direction), n)));
				ray_origin += n*EPS;
			}
			else
			{
				//total internal reflection
				ray_direction = reflected_direction;
				ray_origin -= n*EPS;
			}
		}
		else
		{
			//calculating of outcoming ray using reflection_factor
			ray_direction = normalize((1.0f - reflection_factor)*random_direction + reflection_factor*reflected_direction);
			ray_origin += n*EPS;
		}

		accumulated += mask*polygons[nearest_id].material.emit * polygons[nearest_id].material.color;

		mask *= polygons[nearest_id].material.color;
		mask *= dot(n, ray_direction)*(1.0 - reflection_factor) + reflection_factor;
		mask *= magic_factor*(1.0 - reflection_factor) + reflection_factor;
	}
	accumulator[idx] += accumulated;
	output[idx] = accumulator[idx] / (iteration + 1);
}

void path_tracing(int iteration, float3 * buffer)
{
	if (iteration == 0 || accumulator_d == 0)
	{
		hipFree(accumulator_d);
		hipFree(buffer_d);
		hipMalloc(&accumulator_d, image_size * sizeof(float3));
		hipMemset(accumulator_d, 0, image_size * sizeof(float3));
		hipMalloc(&buffer_d, image_size * sizeof(float3));
	}
	int threads = max_threads;
	int blocks = (image_size + threads - 1) / threads;
	render_kernel << <blocks, threads >> > (
		iteration, //iteration
		rand() * RAND_MAX +  rand(), //seed
		polygons_number, //number of polygons
		polygons_d, //polygons array
		accumulator_d, //pixels accumulator
		buffer_d, //output pixel buffer
		camera, //camera position
		direction //direction from camera to target
		);
	hipMemcpy(buffer, buffer_d, image_size * sizeof(float3), hipMemcpyDeviceToHost);
}
